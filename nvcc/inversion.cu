#include "hip/hip_runtime.h"
#include <stdio.h>
#include <CL/cl.hpp>
#include "hip/hip_runtime.h"

#define cudaCheckErrors() {                                          \
hipError_t e=hipGetLastError();                                 \
if(e!=hipSuccess) {                                              \
printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
exit(0); \
}                                                                 \
}

__host__ __device__ void
print_matrix(float *matrix, float *iden, int dim, int xto, int yto, int xfrom = 0, int yfrom = 0) {
	for (int i = yfrom; i < yto; i++) {
		for (int j = xfrom; j < xto; j++) {
			printf("%2f ", matrix[i * dim + j]);
		}
		printf("\t\t");
		for (int j = xfrom; j < xto; j++) {
			printf("%2f ", iden[i * dim + j]);
		}
		printf("\n");
	}
	printf("\n");
}

__host__ __device__ void print_matrix(float *matrix, float *iden, int dim, int xfrom = 0, int yfrom = 0) {
	print_matrix(matrix, iden, dim, dim, dim, xfrom, yfrom);
}


void openacc_offload(float *matrix, float *iden, int dim) {
#pragma acc data copy(matrix[0:dim * dim], iden[0:dim * dim])
	for (int i = 0; i < dim; i++) {
		if (matrix[i * dim + i] == 0) { // swap lines if 0
			for (int j = i + 1; j < dim; j++) { // find new line
				if (matrix[j * dim + i] != 0) {
#pragma acc parallel loop worker vector//vector_length(32)
					for (int x = i; x < dim; x++) { // swap lines
						matrix[i * dim + x] += matrix[j * dim + x];
						iden[i * dim + x] += iden[j * dim + x];
					}
					break;
				}
			}
		}
		
		//normalize

#pragma acc parallel loop gang worker vector
		for (int x = i + 1; x < dim + i + 1; x++) {
			float factor = matrix[i * dim + i];
			if (x < dim)
				matrix[i * dim + x] /= factor;
			else {
				iden[i * dim + x - dim] /= factor;
			}
		}
#pragma acc serial
		{
			matrix[i * dim + i] = 1;
		};
		
		//gauss
#pragma acc parallel loop gang worker device_type(nvidia)
		for (int y = 0; y < dim; y++) {
			float factor = matrix[y * dim + i];
			if (y != i && factor != 0.0f) {
#pragma acc loop vector
				for (int x = i; x < dim + i + 1; x++) {
					if (x < dim)
						matrix[y * dim + x] -= matrix[i * dim + x] * factor;
					else
						iden[y * dim + x - dim] -= iden[i * dim + x - dim] * factor;
				}
			}
		}
	}
}


__global__ void normalize(float *matrix, float *iden, int iter, int dim) {
	int x = 1 + iter + blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= 2 * dim)
		return;
	
	
	if (x < dim)
		matrix[iter * dim + x] /= matrix[iter * dim + iter];
	else
		iden[iter * dim + x - dim] /= matrix[iter * dim + iter];
}

__global__ void gauss(float *matrix, float *iden, int iter, int dim) {
	int iterations_per_thread = std::ceil((dim + 1.) / 128.);
	int x = 1 + iter + threadIdx.x * iterations_per_thread;
	
	if (x >= 2 * dim)
		return;
	
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (y >= iter) y++;
	float factor = matrix[y * dim + iter];
	
	for (; x <= iter + (threadIdx.x + 1) * iterations_per_thread && x <= iter + dim; x++) {
		if (x < dim)
			matrix[y * dim + x] -= matrix[iter * dim + x] * factor;
		else
			iden[(y - 1) * dim + x] -= iden[(iter - 1) * dim + x] * factor;
	}
	
	matrix[y * dim + iter] = 0;
}

__global__ void gauss_fix(float *matrix, int iter, int dim) {
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (y >= dim || y == iter)
		return;
	
	matrix[y * dim + iter] = 0;
}


void cuda_offload(float *matrix, float *iden, int dim) {
	float *d_A, *d_I;
	
	// setup and copy matrices to gpu
	hipMalloc(&d_A, dim * dim * sizeof(float));
	hipMalloc(&d_I, dim * dim * sizeof(float));
	hipMemcpy(d_A, matrix, dim * dim * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_I, iden, dim * dim * sizeof(float), hipMemcpyHostToDevice);
	cudaCheckErrors();
	
	// setup kernelsizes
	
	int threadsperblock = 128;
	int blocks = std::ceil(static_cast<float>(dim) / threadsperblock);
	
	dim3 norm_block(threadsperblock);
	dim3 norm_grid(blocks);
	
	
	dim3 gauss_block(threadsperblock);
	dim3 gauss_grid(1, dim - 1);
	
	dim3 gauss_fix_block(1, threadsperblock);
	dim3 gauss_fix_grid(1, blocks);
	
	for (int iter = 0; iter < dim; iter++) {
		if (matrix[iter * dim + iter] == 0) { // swap lines if 0 -> divide by 0 is impossible
			for (int j = iter + 1; j < dim; j++) { // find new line
				if (matrix[j * dim + iter] != 0) {
					for (int x = iter; x < dim; x++) { // swap lines
						matrix[iter * dim + x] += matrix[j * dim + x];
						iden[iter * dim + x] += iden[j * dim + x];
					}
					break;
				}
			}
		}
		
		//normalize
		normalize<<<norm_grid, norm_block>>>(d_A, d_I, iter, dim);
		hipDeviceSynchronize();
		cudaCheckErrors();
		matrix[iter * dim + iter] = 1;
		hipMemcpy(&d_A[iter * dim + iter], &matrix[iter * dim + iter], sizeof(float), hipMemcpyHostToDevice);
		//hipMemcpy(iden, d_I, dim * dim * sizeof(float), hipMemcpyDeviceToHost);
		//hipMemcpy(matrix, d_A, dim * dim * sizeof(float), hipMemcpyDeviceToHost);
		//print_matrix(matrix, iden, dim);
		cudaCheckErrors();
		
		//gauss
		gauss<<<gauss_grid, gauss_block>>>(d_A, d_I, iter, dim);
		hipDeviceSynchronize();
		cudaCheckErrors();
		
		gauss_fix<<<gauss_fix_grid, gauss_fix_block>>>(d_A, iter, dim);
		hipDeviceSynchronize();
		//hipMemcpy(iden, d_I, dim * dim * sizeof(float), hipMemcpyDeviceToHost);
		//hipMemcpy(matrix, d_A, dim * dim * sizeof(float), hipMemcpyDeviceToHost);
		//print_matrix(matrix, iden, dim);
		//cudaCheckErrors();
	}
	cudaCheckErrors();
	
	// Copy results back to host
	hipDeviceSynchronize();
	cudaCheckErrors();
	hipMemcpy(iden, d_I, dim * dim * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(matrix, d_A, dim * dim * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_I);
	cudaCheckErrors();
}

