#include "hipblas.h"
#include "hip/hip_runtime.h"

#ifdef dbl
using scalar = double;
#else
using scalar = float;
#endif

#define cudaCheckErrors()                                                                    \
	{                                                                                        \
		hipError_t e = hipGetLastError();                                                  \
		if (e != hipSuccess) {                                                              \
			printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
			exit(0);                                                                         \
		}                                                                                    \
	}

void cublas_offload(scalar *matrix, scalar *iden, int dim) {
	scalar *d_A, *d_I;

	// setup and copy matrices to gpu
	hipMalloc(&d_A, dim * dim * sizeof(scalar));
	hipMalloc(&d_I, dim * dim * sizeof(scalar));
	hipMemcpy(d_A, matrix, dim * dim * sizeof(scalar), hipMemcpyHostToDevice);
	hipMemcpy(d_I, iden, dim * dim * sizeof(scalar), hipMemcpyHostToDevice);
	cudaCheckErrors();

	// setup kernelsizes
	int info[1];
	hipblasHandle_t cu_handle;
	hipblasCreate(&cu_handle);
	scalar *const *matrices = &matrix;
	scalar *const *identities = &iden;

	hipblasSgetrfBatched(cu_handle, dim, matrices, dim, NULL, info, 1);
	hipblasSgetriBatched(cu_handle, dim, matrices, dim, NULL, identities, dim, info, 1);
	cudaCheckErrors();
	hipMemcpy(iden, d_I, dim * dim * sizeof(scalar), hipMemcpyDeviceToHost);
	hipMemcpy(matrix, d_A, dim * dim * sizeof(scalar), hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_I);
	cudaCheckErrors();
}
