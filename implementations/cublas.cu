#include "hipblas.h"
#include "hip/hip_runtime.h"

#ifdef dbl
using scalar = double;
#else
using scalar = float;
#endif


#define cudacall(call)                                                                                                        \
	do {                                                                                                                      \
		hipError_t err = (call);                                                                                             \
		if (hipSuccess != err) {                                                                                             \
			fprintf(stderr, "CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
			hipDeviceReset();                                                                                                \
			exit(EXIT_FAILURE);                                                                                               \
		}                                                                                                                     \
	} while (0)

#define cublascall(call)                                                                                     \
	do {                                                                                                     \
		hipblasStatus_t status = (call);                                                                      \
		if (HIPBLAS_STATUS_SUCCESS != status) {                                                               \
			fprintf(stderr, "CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status); \
			hipDeviceReset();                                                                               \
			exit(EXIT_FAILURE);                                                                              \
		}                                                                                                    \
                                                                                                             \
	} while (0)

void cublas_offload(scalar *A, scalar *I, int dim) {
	auto **As = (scalar **) new scalar *;
	auto **Is = (scalar **) new scalar *;
	scalar **d_As;
	scalar **d_Is;
	scalar *d_A;
	scalar *d_I;

	cudacall(hipMalloc(&d_As, sizeof(scalar *)));
	cudacall(hipMalloc(&d_Is, sizeof(scalar *)));
	cudacall(hipMalloc(&d_A, dim * dim * sizeof(scalar)));
	cudacall(hipMalloc(&d_I, dim * dim * sizeof(scalar)));
	As[0] = d_A;
	Is[0] = d_I;
	cudacall(hipMemcpy(d_As, As, sizeof(scalar *), hipMemcpyHostToDevice));
	cudacall(hipMemcpy(d_Is, Is, sizeof(scalar *), hipMemcpyHostToDevice));
	cudacall(hipMemcpy(d_A, A, dim * dim * sizeof(scalar), hipMemcpyHostToDevice));


	hipblasHandle_t cu_handle;
	cublascall(hipblasCreate(&cu_handle));
	int *pivot_element;
	int *d_info;
	cudacall(hipMalloc(&pivot_element, sizeof(int)));
	cudacall(hipMalloc(&d_info, sizeof(int)));

#ifdef dbl
	cublascall(cublasDgetrfBatched(cu_handle, dim, d_As, dim, pivot_element, d_info, 1));
#else
	cublascall(hipblasSgetrfBatched(cu_handle, dim, d_As, dim, pivot_element, d_info, 1));
#endif

#ifdef dbl
	cublascall(cublasDgetriBatched(cu_handle, dim, (const scalar **) d_As, dim, pivot_element, d_Is, dim, d_info, 1));
#else
	cublascall(hipblasSgetriBatched(cu_handle, dim, (const scalar **) d_As, dim, pivot_element, d_Is, dim, d_info, 1));
#endif
	cudacall(hipMemcpy(I, d_I, dim * dim * sizeof(scalar), hipMemcpyDeviceToHost));

	hipFree(d_As);
	hipFree(d_A);
	hipFree(d_I);
	hipFree(d_Is);
	free(As);
	hipFree(pivot_element);
	hipFree(d_info);
	hipblasDestroy(cu_handle);
}
