#include "hipblas.h"
#include "hip/hip_runtime.h"

#ifdef dbl
using scalar = double;
#else
using scalar = float;
#endif


#define cudacall(call)                                                                                                        \
	do {                                                                                                                      \
		hipError_t err = (call);                                                                                             \
		if (hipSuccess != err) {                                                                                             \
			fprintf(stderr, "CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
			hipDeviceReset();                                                                                                \
			exit(EXIT_FAILURE);                                                                                               \
		}                                                                                                                     \
	} while (0)

#define cublascall(call)                                                                                     \
	do {                                                                                                     \
		hipblasStatus_t status = (call);                                                                      \
		if (HIPBLAS_STATUS_SUCCESS != status) {                                                               \
			fprintf(stderr, "CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status); \
			hipDeviceReset();                                                                               \
			exit(EXIT_FAILURE);                                                                              \
		}                                                                                                    \
                                                                                                             \
	} while (0)

void cublas_offload(scalar *matrix, scalar *result, int dim) {
	scalar **inputs = (scalar **) new scalar *;
	auto **results = (scalar **) new scalar *;
	scalar **d_results;
	scalar *d_result;

	cudacall(hipMalloc(&d_results, sizeof(scalar *)));
	cudacall(hipMalloc(&d_result, dim * dim * sizeof(scalar)));
	results[0] = d_result;
	cudacall(hipMemcpy(d_results, results, sizeof(scalar *), hipMemcpyHostToDevice));

	hipblasHandle_t cu_handle;
	cublascall(hipblasCreate(&cu_handle));

	int *pivot_element;
	int h_info[1];
	int *d_info;

	cudacall(hipMalloc(&pivot_element, sizeof(int)));
	cudacall(hipMalloc(&d_info, sizeof(int)));

	auto **matrices = (scalar **) new scalar *;
	scalar **d_matrices;
	scalar *d_matrix;

	cudacall(hipMalloc(&d_matrices, sizeof(scalar *)));
	cudacall(hipMalloc(&d_matrix, dim * dim * sizeof(scalar)));

	matrices[0] = d_matrix;

	cudacall(hipMemcpy(d_matrices, matrices, sizeof(scalar *), hipMemcpyHostToDevice));
	cudacall(hipMemcpy(d_matrix, matrix, dim * dim * sizeof(scalar), hipMemcpyHostToDevice));

#ifdef dbl
	cublascall(cublasDgetrfBatched(cu_handle, dim, d_matrices, dim, pivot_element, d_info, 1));
#else
	cublascall(hipblasSgetrfBatched(cu_handle, dim, d_matrices, dim, pivot_element, d_info, 1));
#endif
	cudacall(hipMemcpy(h_info, d_info, sizeof(int), hipMemcpyDeviceToHost));
	if (h_info[0] != 0) {
		fprintf(stderr, "Factorization of matrix %d Failed: Matrix may be singular\n", 0);
		hipDeviceReset();
		exit(EXIT_FAILURE);
	}

	scalar **C = (scalar **) new scalar *;
	scalar **C_d, *C_dflat;

	cudacall(hipMalloc(&C_d, sizeof(scalar *)));
	cudacall(hipMalloc(&C_dflat, dim * dim * sizeof(scalar)));
	C[0] = C_dflat;
	cudacall(hipMemcpy(C_d, C, sizeof(scalar *), hipMemcpyHostToDevice));

#ifdef dbl
	cublascall(cublasDgetriBatched(cu_handle, dim, (const scalar **) d_matrices, dim, pivot_element, d_results, dim, d_info, 1));
#else
	cublascall(hipblasSgetriBatched(cu_handle, dim, (const scalar **) d_matrices, dim, pivot_element, d_results, dim, d_info, 1));
#endif
	cudacall(hipMemcpy(h_info, d_info, sizeof(int), hipMemcpyDeviceToHost));
	if (h_info[0] != 0) {
		fprintf(stderr, "Inversion of matrix %d Failed: Matrix may be singular\n", 0);
		hipDeviceReset();
		exit(EXIT_FAILURE);
	}
	cudacall(hipMemcpy(result, d_result, dim * dim * sizeof(scalar), hipMemcpyDeviceToHost));

	hipFree(d_matrices);
	hipFree(d_matrix);
	hipFree(d_result);
	hipFree(d_results);
	free(matrices);
	free(C);
	hipFree(pivot_element);
	hipFree(d_info);
	hipblasDestroy(cu_handle);
}
