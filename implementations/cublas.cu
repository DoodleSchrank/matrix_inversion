#include "hipblas.h"
#include "hip/hip_runtime.h"

#ifdef dbl
using scalar = double;
#else
using scalar = float;
#endif


#define cudacall(call)                                                                                                        \
	do {                                                                                                                      \
		hipError_t err = (call);                                                                                             \
		if (hipSuccess != err) {                                                                                             \
			fprintf(stderr, "CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
			hipDeviceReset();                                                                                                \
			exit(EXIT_FAILURE);                                                                                               \
		}                                                                                                                     \
	} while (0)

#define cublascall(call)                                                                                     \
	do {                                                                                                     \
		hipblasStatus_t status = (call);                                                                      \
		if (HIPBLAS_STATUS_SUCCESS != status) {                                                               \
			fprintf(stderr, "CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status); \
			hipDeviceReset();                                                                               \
			exit(EXIT_FAILURE);                                                                              \
		}                                                                                                    \
                                                                                                             \
	} while (0)

void cublas_offload(scalar *matrix, scalar *result, int dim) {
	scalar **inputs = (scalar **) new scalar *;
	auto **results = (scalar **) new scalar *;
	scalar **d_results;
	scalar *d_result;
	
	cudacall(hipMalloc(&d_results, sizeof(float *)));
	cudacall(hipMalloc(&d_result, dim * dim * sizeof(float)));
	results[0] = d_result;
	cudacall(hipMemcpy(d_results, results, sizeof(float *), hipMemcpyHostToDevice));
	
	hipblasHandle_t cu_handle;
	cublascall(hipblasCreate(&cu_handle));

	int *pivot_element;
	int h_info[1];
	int *d_info;

	cudacall(hipMalloc(&pivot_element, sizeof(int)));
	cudacall(hipMalloc(&d_info, sizeof(int)));

	auto **matrices = (scalar **) new scalar *;
	scalar **d_matrices;
	scalar *d_matrix;

	cudacall(hipMalloc(&d_matrices, sizeof(scalar *)));
	cudacall(hipMalloc(&d_matrix, dim * dim * sizeof(scalar)));

	matrices[0] = d_matrix;

	cudacall(hipMemcpy(d_matrices, matrices, sizeof(scalar *), hipMemcpyHostToDevice));
	cudacall(hipMemcpy(d_matrix, matrix, dim * dim * sizeof(scalar), hipMemcpyHostToDevice));

	cublascall(hipblasSgetrfBatched(cu_handle, dim, d_matrices, dim, pivot_element, d_info, 1));
	cudacall(hipMemcpy(h_info, d_info, sizeof(int), hipMemcpyDeviceToHost));
	if (h_info[0] != 0) {
		fprintf(stderr, "Factorization of matrix %d Failed: Matrix may be singular\n", 0);
		hipDeviceReset();
		exit(EXIT_FAILURE);
	}

	float **C = (float **) new float *;
	float **C_d, *C_dflat;

	cudacall(hipMalloc(&C_d, sizeof(float *)));
	cudacall(hipMalloc(&C_dflat, dim * dim * sizeof(float)));
	C[0] = C_dflat;
	cudacall(hipMemcpy(C_d, C, sizeof(float *), hipMemcpyHostToDevice));


	cublascall(hipblasSgetriBatched(cu_handle, dim, (const scalar **) d_matrices, dim, pivot_element, d_results, dim, d_info, 1));
	cudacall(hipMemcpy(h_info, d_info, sizeof(int), hipMemcpyDeviceToHost));
	if (h_info[0] != 0) {
		fprintf(stderr, "Inversion of matrix %d Failed: Matrix may be singular\n", 0);
		hipDeviceReset();
		exit(EXIT_FAILURE);
	}
	cudacall(hipMemcpy(result, d_result, dim * dim * sizeof(scalar), hipMemcpyDeviceToHost));

	hipFree(d_matrices);
	hipFree(d_matrix);
	hipFree(d_result);
	hipFree(d_results);
	free(matrices);
	free(C);
	hipFree(pivot_element);
	hipFree(d_info);
	hipblasDestroy(cu_handle);
}