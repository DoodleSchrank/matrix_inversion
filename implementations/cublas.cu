#include "hipblas.h"
#include "hip/hip_runtime.h"

#ifdef dbl
using scalar = double;
#else
using scalar = float;
#endif


#define cudacall(call)                                                                                                        \
	do {                                                                                                                      \
		hipError_t err = (call);                                                                                             \
		if (hipSuccess != err) {                                                                                             \
			fprintf(stderr, "CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
			hipDeviceReset();                                                                                                \
			exit(EXIT_FAILURE);                                                                                               \
		}                                                                                                                     \
	} while (0)

#define cublascall(call)                                                                                     \
	do {                                                                                                     \
		hipblasStatus_t status = (call);                                                                      \
		if (HIPBLAS_STATUS_SUCCESS != status) {                                                               \
			fprintf(stderr, "CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status); \
			hipDeviceReset();                                                                               \
			exit(EXIT_FAILURE);                                                                              \
		}                                                                                                    \
                                                                                                             \
	} while (0)

void cublas_offload(float *A, float *I, int dim) {
	auto **As = (float **) new float *;
	auto **Is = (float **) new float *;
	float **d_As;
	float **d_Is;
	float *d_A;
	float *d_I;

	cudacall(hipMalloc(&d_As, sizeof(float *)));
	cudacall(hipMalloc(&d_Is, sizeof(float *)));
	cudacall(hipMalloc(&d_A, dim * dim * sizeof(float)));
	cudacall(hipMalloc(&d_I, dim * dim * sizeof(float)));
	As[0] = d_A;
	Is[0] = d_I;
	cudacall(hipMemcpy(d_As, As, sizeof(float *), hipMemcpyHostToDevice));
	cudacall(hipMemcpy(d_Is, Is, sizeof(float *), hipMemcpyHostToDevice));
	cudacall(hipMemcpy(d_A, A, dim * dim * sizeof(float), hipMemcpyHostToDevice));


	hipblasHandle_t cu_handle;
	cublascall(hipblasCreate(&cu_handle));
	int *pivot_element;
	int *d_info;
	cudacall(hipMalloc(&pivot_element, sizeof(int)));
	cudacall(hipMalloc(&d_info, sizeof(int)));

#ifdef dbl
	cublascall(cublasDgetrfBatched(cu_handle, dim, d_As, dim, pivot_element, d_info, 1));
#else
	cublascall(hipblasSgetrfBatched(cu_handle, dim, d_As, dim, pivot_element, d_info, 1));
#endif

#ifdef dbl
	cublascall(cublasDgetriBatched(cu_handle, dim, (const float **) d_As, dim, pivot_element, d_Is, dim, d_info, 1));
#else
	cublascall(hipblasSgetriBatched(cu_handle, dim, (const float **) d_As, dim, pivot_element, d_Is, dim, d_info, 1));
#endif
	cudacall(hipMemcpy(I, d_I, dim * dim * sizeof(float), hipMemcpyDeviceToHost));

	hipFree(d_As);
	hipFree(d_A);
	hipFree(d_I);
	hipFree(d_Is);
	free(As);
	hipFree(pivot_element);
	hipFree(d_info);
	hipblasDestroy(cu_handle);
}
