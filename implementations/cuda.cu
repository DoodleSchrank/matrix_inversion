#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef dbl
using scalar = double;
#else
using scalar = float;
#endif

#define cudacall(call)                                                                                                        \
	do {                                                                                                                      \
		hipError_t err = (call);                                                                                             \
		if (hipSuccess != err) {                                                                                             \
			fprintf(stderr, "CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
			hipDeviceReset();                                                                                                \
			exit(EXIT_FAILURE);                                                                                               \
		}                                                                                                                     \
	} while (0)

__global__ void finddiagonal(scalar *A, scalar *I, int iter, int dim) {
	int column = threadIdx.x;
	__shared__ int newline = 0;
	if (column == 0) {
		for (int row = iter + 1; row < dim; row++) {// find new line
			if (A[row * dim + iter] != 0) {
				newline = row;
			}
		}
	}
	__syncthreads();

	for (int i = column; i < 2 * dim; i += blockDim.x) {
		if (i < dim) {
			A[iter * dim + i] += A[newline * dim + i];
		} else {
			I[iter * dim + i - dim] += I[newline * dim + i - dim];
		}
	}
}

__global__ void normalize(scalar *A, scalar *I, int iter, int dim) {
	__shared__ scalar diag_elem = A[iter * dim + iter];
	__syncthreads();

	int column = threadIdx.x;

	for (int i = column; i < dim + iter + 1; i += blockDim.x) {
		if (i < dim) {
			A[iter * dim + i] /= diag_elem;
		} else if (i < 2 * dim) {
			I[iter * dim + i - dim] /= diag_elem;
		}
	}
}

__global__ void gauss(scalar *A, scalar *I, int iter, int dim) {
	int column = 1 + iter + blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (column >= 2 * dim || row == iter)
		return;

	scalar factor = A[row * dim + iter];

	for (int i = column; i < dim + iter + 1; i += blockDim.x) {
		if (i < dim) {
			A[row * dim + i] -= A[iter * dim + i] * factor;
		} else if (i < 2 * dim) {
			I[row * dim + i - dim] -= I[iter * dim + i - dim] * factor;
		}
	}
}

__global__ void gauss_fix(scalar *A, int iter, int dim) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= dim || row == iter)
		return;
	for (int i = row; i < dim; i += blockDim.x) {
		A[i * dim + iter] = 0;
	}
}

void cuda_offload(scalar *A, scalar *I, int dim) {
	scalar *d_A, *d_I;

	// setup and copy matrices to gpu
	cudacall(hipMalloc(&d_A, dim * dim * sizeof(scalar)));
	cudacall(hipMalloc(&d_I, dim * dim * sizeof(scalar)));
	cudacall(hipMemcpy(d_A, A, dim * dim * sizeof(scalar), hipMemcpyHostToDevice));
	cudacall(hipMemcpy(d_I, I, dim * dim * sizeof(scalar), hipMemcpyHostToDevice));

	// setup kernel sizes
	struct hipDeviceProp_t properties;
	cudacall(hipGetDeviceProperties(&properties, 0));

	int threads = min(2 * dim, properties.maxThreadsPerBlock);
	dim3 norm_block(threads);
	dim3 norm_grid(1);

	threads = min(2 * dim, properties.maxThreadsPerBlock);
	dim3 gauss_block(threads);
	dim3 gauss_grid(1, dim);

	for (int iter = 0; iter < dim; iter++) {
		// swap lines if 0 -> divide by 0 is not allowed
		if (A[iter * dim + iter] == 0) {
			finddiagonal<<<norm_grid, norm_block>>>(d_A, d_I, iter, dim);
		}

		//normalize
		normalize<<<norm_grid, norm_block>>>(d_A, d_I, iter, dim);
		cudacall(hipDeviceSynchronize());

		//gauss
		gauss<<<gauss_grid, gauss_block>>>(d_A, d_I, iter, dim);
		cudacall(hipDeviceSynchronize());
		gauss_fix<<<norm_grid, norm_block>>>(d_A, iter, dim);
		cudacall(hipDeviceSynchronize());
	}

	// Copy results back to host
	cudacall(hipDeviceSynchronize());
	cudacall(hipMemcpy(I, d_I, dim * dim * sizeof(scalar), hipMemcpyDeviceToHost));
	cudacall(hipMemcpy(A, d_A, dim * dim * sizeof(scalar), hipMemcpyDeviceToHost));
	cudacall(hipFree(d_A));
	cudacall(hipFree(d_I));
}
