#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"

#ifdef dbl
using scalar = double;
#else
using scalar = float;
#endif

#define cudaCheckErrors()                                                                    \
	{                                                                                        \
		hipError_t e = hipGetLastError();                                                  \
		if (e != hipSuccess) {                                                              \
			printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
			exit(0);                                                                         \
		}                                                                                    \
	}


__global__ void finddiagonal(scalar *matrix, scalar *iden, int iter, int dim) {
	int x = threadIdx.x;
	__shared__ int newline = 0;
	if (x == 0) {
		for (int j = iter + 1; j < dim; j++) {// find new line
			if (matrix[j * dim + iter] != 0) {
				newline = j;
			}
		}
	}
	__syncthreads();

	for (int i = x; i < 2 * dim; i += blockDim.x) {
		if (i < dim) {
			scalar temp = matrix[iter * dim + i];
			matrix[iter * dim + i] = matrix[newline * dim + i];
			matrix[newline * dim + i] = temp;
		} else {
			scalar temp = iden[iter * dim + i - dim];
			iden[iter * dim + i - dim] = iden[newline * dim + i - dim];
			iden[newline * dim + i - dim] = temp;
		}
	}
}

__global__ void normalize(scalar *matrix, scalar *iden, int iter, int dim) {
	__shared__ scalar diag_elem = matrix[iter * dim + iter];
	__syncthreads();

	int x = threadIdx.x;

	for (int i = x; i < 2 * dim; i += blockDim.x) {
		if (i < dim)
			matrix[iter * dim + i] /= diag_elem;
		else
			iden[iter * dim + i - dim] /= diag_elem;
	}
}


__global__ void gauss(scalar *matrix, scalar *iden, int iter, int dim) {
	int x = 1 + iter + blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= 2 * dim || y == iter)
		return;

	scalar factor = matrix[y * dim + iter];


	if (x < dim)
		matrix[y * dim + x] -= matrix[iter * dim + x] * factor;
	else
		iden[y * dim + x - dim] -= iden[iter * dim + x - dim] * factor;
}

__global__ void gauss_fix(scalar *matrix, int iter, int dim) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x >= dim || x == iter)
		return;

	matrix[x * dim + iter] = 0;
}


void cuda_offload(scalar *matrix, scalar *iden, int dim) {
	scalar *d_A, *d_I;

	// setup and copy matrices to gpu
	hipMalloc(&d_A, dim * dim * sizeof(scalar));
	hipMalloc(&d_I, dim * dim * sizeof(scalar));
	hipMemcpy(d_A, matrix, dim * dim * sizeof(scalar), hipMemcpyHostToDevice);
	hipMemcpy(d_I, iden, dim * dim * sizeof(scalar), hipMemcpyHostToDevice);
	cudaCheckErrors();

	// setup kernelsizes

	struct hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);

	int row_parts = 1;
	int threads = min(2 * dim, properties.maxThreadsPerBlock);
	dim3 norm_block(threads);
	dim3 norm_grid(row_parts);
	row_parts = (2 * dim > properties.maxThreadsPerBlock) ? std::ceil(2. * dim / properties.maxThreadsPerBlock) : 1;
	threads = std::ceil(2. * dim / row_parts);
	dim3 gauss_block(threads);
	dim3 gauss_grid(row_parts, dim);

	for (int iter = 0; iter < dim; iter++) {
		if (matrix[iter * dim + iter] == 0) {// swap lines if 0 -> divide by 0 is impossible
			finddiagonal<<<norm_grid, norm_block>>>(matrix, iden, iter, dim);
		}

		//normalize
		normalize<<<norm_grid, norm_block>>>(d_A, d_I, iter, dim);
		hipDeviceSynchronize();
		cudaCheckErrors();

		//gauss
		gauss<<<gauss_grid, gauss_block>>>(d_A, d_I, iter, dim);
		hipDeviceSynchronize();
		gauss_fix<<<norm_grid, norm_block>>>(d_A, iter, dim);
		hipDeviceSynchronize();
		cudaCheckErrors();
	}
	cudaCheckErrors();

	// Copy results back to host
	hipDeviceSynchronize();
	cudaCheckErrors();
	hipMemcpy(iden, d_I, dim * dim * sizeof(scalar), hipMemcpyDeviceToHost);
	hipMemcpy(matrix, d_A, dim * dim * sizeof(scalar), hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_I);
	cudaCheckErrors();
}
