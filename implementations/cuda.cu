#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"

#ifdef dbl
using scalar = double;
#else
using scalar = float;
#endif

#define cudaCheckErrors()                                                                    \
	{                                                                                        \
		hipError_t e = hipGetLastError();                                                  \
		if (e != hipSuccess) {                                                              \
			printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
			exit(0);                                                                         \
		}                                                                                    \
	}


__global__ void finddiagonal(scalar *A, scalar *I, int iter, int dim) {
	int column = threadIdx.x;
	__shared__ int newline = 0;
	if (column == 0) {
		for (int row = iter + 1; row < dim; row++) {// find new line
			if (A[row * dim + iter] != 0) {
				newline = row;
			}
		}
	}
	__syncthreads();

	for (int i = column; i < 2 * dim; i += blockDim.x) {
		if (i < dim) {
			A[iter * dim + i] += A[newline * dim + i];
		} else {
			I[iter * dim + i - dim] += I[newline * dim + i - dim];
		}
	}
}

__global__ void normalize(scalar *A, scalar *I, int iter, int dim) {
	__shared__ scalar diag_elem = A[iter * dim + iter];
	__syncthreads();

	int column = threadIdx.x;

	for (int i = column; i < dim + iter + 1; i += blockDim.x) {
		if (i < dim) {
			A[iter * dim + i] /= diag_elem;
		} else if (i < 2 * dim) {
			I[iter * dim + i - dim] /= diag_elem;
		}
	}
}


__global__ void gauss(scalar *A, scalar *I, int iter, int dim) {
	int column = 1 + iter + blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (column >= 2 * dim || row == iter)
		return;

	scalar factor = A[row * dim + iter];

	for (int i = column; i < dim + iter + 1; i += blockDim.x) {
		if (i < dim) {
			A[row * dim + i] -= A[iter * dim + i] * factor;
		} else  if (i < 2 * dim) {
			I[row * dim + i - dim] -= I[iter * dim + i - dim] * factor;
		}
	}
}

__global__ void gauss_fix(scalar *A, int iter, int dim) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= dim || row == iter)
		return;
	for (int i = row; i < dim; i += blockDim.x) {
		A[i * dim + iter] = 0;
	}
}


void cuda_offload(scalar *A, scalar *I, int dim) {
	scalar *d_A, *d_I;

	// setup and copy matrices to gpu
	hipMalloc(&d_A, dim * dim * sizeof(scalar));
	hipMalloc(&d_I, dim * dim * sizeof(scalar));
	hipMemcpy(d_A, A, dim * dim * sizeof(scalar), hipMemcpyHostToDevice);
	hipMemcpy(d_I, I, dim * dim * sizeof(scalar), hipMemcpyHostToDevice);
	cudaCheckErrors();

	// setup kernel sizes

	struct hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);

	int threads = min(2 * dim, properties.maxThreadsPerBlock);
	dim3 norm_block(threads);
	dim3 norm_grid(1);
	
	threads = min(2 * dim, properties.maxThreadsPerBlock);
	dim3 gauss_block(threads);
	dim3 gauss_grid(1, dim);

	for (int iter = 0; iter < dim; iter++) {
		// swap lines if 0 -> divide by 0 is not allowed
		if (A[iter * dim + iter] == 0) {
			finddiagonal<<<norm_grid, norm_block>>>(A, I, iter, dim);
		}

		//normalize
		normalize<<<norm_grid, norm_block>>>(d_A, d_I, iter, dim);
		hipDeviceSynchronize();
		cudaCheckErrors();

		//gauss
		gauss<<<gauss_grid, gauss_block>>>(d_A, d_I, iter, dim);
		hipDeviceSynchronize();
		gauss_fix<<<norm_grid, norm_block>>>(d_A, iter, dim);
		hipDeviceSynchronize();
		cudaCheckErrors();
	}
	cudaCheckErrors();

	// Copy results back to host
	hipDeviceSynchronize();
	cudaCheckErrors();
	hipMemcpy(I, d_I, dim * dim * sizeof(scalar), hipMemcpyDeviceToHost);
	hipMemcpy(A, d_A, dim * dim * sizeof(scalar), hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_I);
	cudaCheckErrors();
}
